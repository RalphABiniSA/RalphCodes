
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#define ROWS 400
#define COLS 400
#define EPSILON 1e-6

__device__ double atomicMaxDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void jacobi_method(double* plate, double* next_plate, double* max_diff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i < ROWS - 1 && j < COLS - 1) {
        next_plate[i * COLS + j] = (plate[(i-1) * COLS + j] + plate[(i+1) * COLS + j] + plate[i * COLS + (j-1)] + plate[i * COLS + (j+1)]) / 4.0;
        double diff = fabs(next_plate[i * COLS + j] - plate[i * COLS + j]);
        if (diff > *max_diff) {
            atomicMaxDouble(max_diff, diff);
        }
    }
}



int main() {
    double* plate;
    double* next_plate;
    double* max_diff;
    double* d_plate;
    double* d_next_plate;
    double* d_max_diff;
    int iterations = 0;
    
    //Tempo: inicio
    double secs = 0.0;
    struct timeval start, stop;
    gettimeofday(&start, NULL);
    
    plate = (double*)malloc(ROWS * COLS * sizeof(double));
    next_plate = (double*)malloc(ROWS * COLS * sizeof(double));
    max_diff = (double*)malloc(sizeof(double));

    // Inicialização da placa com valores iniciais
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            plate[i * COLS + j] = 0.0;
        }
    }

    // Definir condições de contorno
    for (int i = 0; i < ROWS; i++) {
        plate[i * COLS] = 100.0;  // Temperatura fixa na borda esquerda
        plate[i * COLS + COLS - 1] = 0.0;  // Temperatura fixa na borda direita
    }

    hipMalloc((void**)&d_plate, ROWS * COLS * sizeof(double));
    hipMalloc((void**)&d_next_plate, ROWS * COLS * sizeof(double));
    hipMalloc((void**)&d_max_diff, sizeof(double));

    hipMemcpy(d_plate, plate, ROWS * COLS * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((ROWS - 2 + blockDim.x - 1) / blockDim.x, (COLS - 2 + blockDim.y - 1) / blockDim.y);

    *max_diff = EPSILON + 1;

    while (*max_diff > EPSILON) {
        *max_diff = 0;

        jacobi_method<<<gridDim, blockDim>>>(d_plate, d_next_plate, d_max_diff);
        hipDeviceSynchronize();

        hipMemcpy(next_plate, d_next_plate, ROWS * COLS * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(max_diff, d_max_diff, sizeof(double), hipMemcpyDeviceToHost);

        for (int i = 1; i < ROWS - 1; i++) {
            for (int j = 1; j < COLS - 1; j++) {
                plate[i * COLS + j] = next_plate[i * COLS + j];
            }
        }

        iterations++;
    }

    printf("Converged after %d iterations\n", iterations);

    //Tempo: final
    gettimeofday(&stop, NULL);
    secs = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);
    printf("Tempo: %lf\n", secs);

    free(plate);
    free(next_plate);
    free(max_diff);
    hipFree(d_plate);
    hipFree(d_next_plate);
    hipFree(d_max_diff);

    return 0;
}
